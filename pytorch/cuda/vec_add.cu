// nvcc vector_add.cu -o vector_add
// ./vector_add


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vector_add(float *a, float *b, float *c, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

int main() {
    int n = 1024;
    float *a, *b, *c;
    float *d_a, *d_b, *d_c;

    // Allocate host memory
    a = (float*)malloc(n * sizeof(float));
    b = (float*)malloc(n * sizeof(float));
    c = (float*)malloc(n * sizeof(float));

    // Initialize host arrays
    for (int i = 0; i < n; ++i) {
        a[i] = i;
        b[i] = i * 2;
    }

    // Allocate device memory
    hipMalloc(&d_a, n * sizeof(float));
    hipMalloc(&d_b, n * sizeof(float));
    hipMalloc(&d_c, n * sizeof(float));

    // Copy host arrays to device
    hipMemcpy(d_a, a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(float), hipMemcpyHostToDevice);

    // Define grid and block sizes
    int threads_per_block = 256;
    int blocks_per_grid = (n + threads_per_block - 1) / threads_per_block;

    // Launch the kernel
    vector_add<<<blocks_per_grid, threads_per_block>>>(d_a, d_b, d_c, n);

    // Copy the result back to the host
    hipMemcpy(c, d_c, n * sizeof(float), hipMemcpyDeviceToHost);

    // Print the result
    for (int i = 0; i < 10; ++i) {
        printf("c[%d] = %f\n", i, c[i]);
    }

    // Free memory
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
